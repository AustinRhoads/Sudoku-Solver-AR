#include "hip/hip_runtime.h"
// Copyright 2017 James Bendig. See the COPYRIGHT file at the top-level
// directory of this distribution.
//
// Licensed under:
//   the MIT license
//     <LICENSE-MIT or https://opensource.org/licenses/MIT>
//   or the Apache License, Version 2.0
//     <LICENSE-APACHE or https://www.apache.org/licenses/LICENSE-2.0>,
// at your option. This file may not be copied, modified, or distributed
// except according to those terms.

#include <iostream>
#include <numeric>
#include <vector>
#include <cassert>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "NeuralNetworkData.h"


static const char* TRAINING_DATA_FILE_PATH = "training.dat";

//CUDA block and thread counts here were manually tuned for a GTX 660.
static const unsigned int PROCESS_NEURON_WEIGHTS_BLOCK_COUNT = 80;
static const unsigned int PROCESS_NEURON_WEIGHTS_THREAD_COUNT = 128;
static const unsigned int UPDATE_OUTPUT_LAYER_BLOCK_COUNT = 80;
static const unsigned int UPDATE_OUTPUT_LAYER_THREAD_COUNT = 512;
static const unsigned int UPDATE_HIDDEN_LAYER_BLOCK_COUNT = 80;
static const unsigned int UPDATE_HIDDEN_LAYER_THREAD_COUNT = 128;

__global__ void ProcessNeuronWeights(float* inputValues,float* inputWeights,float* output,unsigned int neuronCount,unsigned int weightCount)
{
	const unsigned int localIndex = threadIdx.x;

	__shared__ float localSums[PROCESS_NEURON_WEIGHTS_THREAD_COUNT];
	for(unsigned int neuronIndex = blockIdx.x;neuronIndex < neuronCount;neuronIndex += gridDim.x)
	{
		//Compute sum for the current neuron in parallel.
		float sum = 0.0f;
		unsigned int inputWeightsBase = neuronIndex * weightCount;
		for(unsigned int x = localIndex;x < weightCount;x += blockDim.x)
		{
			sum += inputValues[x] * inputWeights[inputWeightsBase + x];
		}
		localSums[localIndex] = sum;
		__syncthreads();

		//Compute total sum for current neuron in parallel.
		for(unsigned int x = PROCESS_NEURON_WEIGHTS_THREAD_COUNT / 2;x > 0;x >>= 1)
		{
			if(localIndex < x)
				localSums[localIndex] += localSums[localIndex + x];
			__syncthreads();
		}

		//Store final result for current neuron.
		if(localIndex == 0)
			output[neuronIndex] = 1.0f / (1.0f + exp(-localSums[0]));
	}
}

__global__ void UpdateOutputLayer(float* previousLayerOutput,float* layerOutput,float* expectedOutput,float* layerWeights,float* layerErrors,unsigned int neuronCount,unsigned int weightCount)
{
	for(unsigned int neuronIndex = blockIdx.x;neuronIndex < neuronCount;neuronIndex += gridDim.x)
	{
		const float error = (expectedOutput[neuronIndex] - layerOutput[neuronIndex]) * layerOutput[neuronIndex] * (1.0f - layerOutput[neuronIndex]);
		if(threadIdx.x == 0)
			layerErrors[neuronIndex] = error;

		//Compute weight changes for the current neuron in parallel.
		const float multiplier = 0.005f * error;
		const unsigned int weightsBase = neuronIndex * weightCount;
		for(unsigned int x = threadIdx.x;x < weightCount;x += blockDim.x)
		{
			layerWeights[weightsBase + x] += previousLayerOutput[x] * multiplier;
		}
	}
}

__global__ void UpdateHiddenLayer(float* previousLayerOutput,float* layerOutput,float* layerWeights,float* nextLayerWeights,float* layerErrors,float* nextLayerErrors,unsigned int neuronCount,unsigned int nextLayerNeuronCount,unsigned int weightCount)
{
	__shared__ float sharedErrors[UPDATE_HIDDEN_LAYER_THREAD_COUNT];
	for(unsigned int neuronIndex = blockIdx.x;neuronIndex < neuronCount;neuronIndex += gridDim.x)
	{
		float error = 0.0f;
		for(unsigned int x = threadIdx.x;x < nextLayerNeuronCount;x += blockDim.x)
		{
			error += nextLayerErrors[x] * nextLayerWeights[x * nextLayerNeuronCount + neuronIndex];
		}
		sharedErrors[threadIdx.x] = error;
		__syncthreads();

		for(unsigned int x = UPDATE_HIDDEN_LAYER_THREAD_COUNT / 2;x > 0;x >>= 1)
		{
			if(threadIdx.x < x)
				sharedErrors[threadIdx.x] += sharedErrors[threadIdx.x + x];
			__syncthreads();
		}

		if(threadIdx.x == 0)
		{
			float finalError = sharedErrors[0];
			finalError *= layerOutput[neuronIndex] * (1.0f - layerOutput[neuronIndex]);
			layerErrors[neuronIndex] = finalError;
			sharedErrors[0] = finalError;
		}
		__syncthreads();

		const float multiplier = 0.005f * sharedErrors[0];
		const unsigned int weightsBase = neuronIndex * weightCount;
		for(unsigned int x = threadIdx.x;x < weightCount;x += blockDim.x)
		{
			layerWeights[weightsBase + x] += previousLayerOutput[x] * multiplier;
		}
	}
}

static void CheckCudaError(const hipError_t err)
{
	if(err == hipSuccess)
		return;

	std::cerr << "Got CUDA error: " << hipGetErrorString(err) << std::endl;
	std::abort();
}

static unsigned long long Milliseconds()
{
	struct timespec tp;
	clock_gettime(CLOCK_MONOTONIC,&tp);
	return static_cast<unsigned long long>(tp.tv_sec) * 1000 +
		   static_cast<unsigned long long>(tp.tv_nsec) / 1000000;
}

static void FetchWeightsFromGPU(const std::vector<float*>& deviceLayerWeights,NeuralNetworkData& nnData)
{
	assert(nnData.layers.size() == deviceLayerWeights.size());

	for(unsigned int l = 0;l < nnData.layers.size();l++)
	{
		Layer& layer = nnData.layers[l];
		assert(!layer.empty());
		const unsigned int neuronSize = layer[0].size();

		for(unsigned int x = 0;x < layer.size();x++)
		{
			const hipError_t err = hipMemcpy(&layer[x][0],&deviceLayerWeights[l][x * neuronSize],neuronSize * sizeof(float),hipMemcpyDeviceToHost);
			CheckCudaError(err);
		}
	}
}

int main(int argc,char* argv[])
{
	//Load existing neural network from file to resume with.
	NeuralNetworkData nnData;
	if(!nnData.LoadFromBinary(TRAINING_DATA_FILE_PATH))
	{
		std::cerr << "Could not load training data." << std::endl;
		return -1;
	}

	hipError_t err = hipSuccess;

	//Query devices for debug reasons. Set selected device manually for now.
	//to a display.
	std::cout << "CUDA Devices" << std::endl;
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	for(int x = 0;x < deviceCount;x++)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp,x);
		std::cout << "[" << x << "]: " << deviceProp.name << std::endl;
	}
	const unsigned int device = 0;
	std::cout << "Using device " << device << std::endl;
	hipSetDevice(device);

	//Make room on the GPU.
	std::vector<float*> deviceTrainingInputs;
	std::vector<float*> deviceExpectedOutputs;
	for(unsigned int x = 0;x < nnData.trainingData.size();x++)
	{
		float* deviceInput = nullptr;
		err = hipMalloc(reinterpret_cast<void**>(&deviceInput),nnData.layers[0][0].size() * sizeof(float));
		CheckCudaError(err);
		deviceTrainingInputs.push_back(deviceInput);

		float* deviceExpected = nullptr;
		err = hipMalloc(reinterpret_cast<void**>(&deviceExpected),nnData.outputChoices.size() * sizeof(float));
		CheckCudaError(err);
		deviceExpectedOutputs.push_back(deviceExpected);
	}

	std::vector<float*> deviceLayerWeights;
	std::vector<float*> deviceLayerOutputs;
	std::vector<float*> deviceLayerErrors;
	for(unsigned int x = 0;x < nnData.layers.size();x++)
	{
		const Layer& layer = nnData.layers[x];

		float* deviceWeights = nullptr;
		err = hipMalloc(reinterpret_cast<void**>(&deviceWeights),layer.size() * layer[0].size() * sizeof(float));
		CheckCudaError(err);
		deviceLayerWeights.push_back(deviceWeights);

		float* deviceOutput = nullptr;
		err = hipMalloc(reinterpret_cast<void**>(&deviceOutput),nnData.layerOutputs[x].size() * sizeof(float));
		CheckCudaError(err);
		deviceLayerOutputs.push_back(deviceOutput);

		float* deviceErrors = nullptr;
		err = hipMalloc(reinterpret_cast<void**>(&deviceErrors),nnData.layerOutputs[x].size() * sizeof(float));
		CheckCudaError(err);
		deviceLayerErrors.push_back(deviceErrors);
	}

	//Transfer data to GPU.
	for(unsigned int x = 0;x < nnData.trainingData.size();x++)
	{
		err = hipMemcpy(deviceTrainingInputs[x],&nnData.trainingData[x].first[0],nnData.layers[0][0].size() * sizeof(float),hipMemcpyHostToDevice);
		CheckCudaError(err);

		AlignedVector expectedOutput;
		ExpectedOutput(nnData.outputChoices,nnData.trainingData[x].second,expectedOutput);
		err = hipMemcpy(deviceExpectedOutputs[x],&expectedOutput[0],nnData.outputChoices.size() * sizeof(float),hipMemcpyHostToDevice);
		CheckCudaError(err);
	}

	for(unsigned int x = 0;x < nnData.layers.size();x++)
	{
		const std::vector<AlignedVector>& layer = nnData.layers[x];
		const unsigned int spanSize = layer[0].size() * sizeof(float);
		for(unsigned int y = 0;y < layer.size();y++)
		{
			err = hipMemcpy(&deviceLayerWeights[x][y * layer[0].size()],&layer[y][0],spanSize,hipMemcpyHostToDevice);
			CheckCudaError(err);
		}

		err = hipMemcpy(deviceLayerOutputs[x],&nnData.layerOutputs[x][0],nnData.layerOutputs[x].size() * sizeof(float),hipMemcpyHostToDevice);
		CheckCudaError(err);
	}

	//Run kernels.
	std::vector<float> outputErrors(nnData.layers.back().size());
	for(unsigned int x = 0;x < 1001;x++)
	{
		const auto startMS = Milliseconds();
		float totalError = 0.0f;
		for(unsigned int y = 0;y < nnData.trainingData.size();y++)
		{
			//Perform forward pass.
			for(unsigned int l = 0;l < nnData.layers.size();l++)
			{
				const unsigned int neuronCount = nnData.layers[l].size();
				const unsigned int weightCount = nnData.layers[l][0].size();
				float* inputValues = nullptr;
				if(l == 0)
					inputValues = deviceTrainingInputs[y];
				else
					inputValues = deviceLayerOutputs[l - 1];
				ProcessNeuronWeights<<<PROCESS_NEURON_WEIGHTS_BLOCK_COUNT,PROCESS_NEURON_WEIGHTS_THREAD_COUNT>>>(inputValues,
																												 deviceLayerWeights[l],
																												 deviceLayerOutputs[l],
																												 neuronCount,
																												 weightCount);
				err = hipGetLastError();
				CheckCudaError(err);
			}

			//Correct output weights.
			UpdateOutputLayer<<<UPDATE_OUTPUT_LAYER_BLOCK_COUNT,UPDATE_OUTPUT_LAYER_THREAD_COUNT>>>(deviceLayerOutputs[deviceLayerOutputs.size() - 2],
																									deviceLayerOutputs.back(),
																									deviceExpectedOutputs[y],
																									deviceLayerWeights.back(),
																									deviceLayerErrors.back(),
																									nnData.layers.back().size(),
																									nnData.layers.back()[0].size());
			err = hipGetLastError();
			CheckCudaError(err);

			//Correct hidden layers' weights.
			for(int l = nnData.layers.size() - 2;l >= 0;l--)
			{
				float* previousLayerOutput = nullptr;
				if(l == 0)
					previousLayerOutput = deviceTrainingInputs[y];
				else
					previousLayerOutput = deviceLayerOutputs[l - 1];
				UpdateHiddenLayer<<<UPDATE_HIDDEN_LAYER_BLOCK_COUNT,UPDATE_HIDDEN_LAYER_THREAD_COUNT>>>(previousLayerOutput,
																										deviceLayerOutputs[l],
																										deviceLayerWeights[l],
																										deviceLayerWeights[l + 1],
																										deviceLayerErrors[l],
																										deviceLayerErrors[l + 1],
																										nnData.layers[l].size(),
																										nnData.layers[l + 1].size(),
																										nnData.layers[l][0].size());
				err = hipGetLastError();
				CheckCudaError(err);
			}

			//Calculate output error.
			err = hipMemcpy(&outputErrors[0],deviceLayerErrors.back(),outputErrors.size() * sizeof(float),hipMemcpyDeviceToHost);
			CheckCudaError(err);
			for(const float error : outputErrors)
			{
				totalError += fabsf(error);
			}
		}

		std::cout << "Training " << x << " took " << (Milliseconds() - startMS) << " ms with error " << totalError << std::endl;

		if(totalError < 1.0f || ((x % 100) == 0 && x != 0))
		{
			FetchWeightsFromGPU(deviceLayerWeights,nnData);
			nnData.SaveAsBinary(TRAINING_DATA_FILE_PATH);
			std::cout << "Saved." << std::endl;
		}
	}

	return 0;
}

